
#include <iostream>
#include <vector>
#include <fstream>

#include <hip/hip_runtime.h>

#include "kernel.cu"

#ifdef _WIN32
#elif __APPLE__
#elif __linux__
    #include <CL/opencl.hpp>
#endif


#define KERNEL_FILE "cuda/kernel.cu"
#define KERNEL_FUNCTION "vector_add"

//https://github.com/zchee/cuda-sample/blob/master/1_Utilities/deviceQuery/deviceQuery.cpp

class ModuleCuda {

private:
    int deviceCount = 0;

    int64_t make_cuda_benchmark(const int device, unsigned matSize, unsigned testIter, const float *matA, const float *matB, float *matR) {

        hipSetDevice(device);

        auto s = std::chrono::high_resolution_clock::now();

        // allocate and set device memory
        unsigned *d_testIter;
        float *d_matA;
        float *d_matB;
        float *d_matR;
        hipMalloc((void**)&d_testIter, sizeof(unsigned));
        hipMalloc((void**)&d_matA, sizeof(float) * matSize);
        hipMalloc((void**)&d_matB, sizeof(float) * matSize);
        hipMalloc((void**)&d_matR, sizeof(float) * matSize);

        // copy data to the device
        hipMemcpy(d_testIter, testIter, sizeof(unsigned), hipMemcpyHostToDevice);
        hipMemcpy(d_matA, matA, sizeof(float) * matSize, hipMemcpyHostToDevice);
        hipMemcpy(d_matB, matB, sizeof(float) * matSize, hipMemcpyHostToDevice);
        hipMemcpy(d_matR, matR, sizeof(float) * matSize, hipMemcpyHostToDevice);
        
        // run benchmark
        int blockSize = 128;
        int gridSize = matSize / blockSize;
        kernel <<<gridSize, blockSize>>>(d_testIter, d_matA, d_matB, d_matR);

        hipMemcpy(testIter, d_testIter, sizeof(unsigned), hipMemcpyDeviceToHost);

        auto e = std::chrono::high_resolution_clock::now();
        auto t = std::chrono::duration_cast<std::chrono::microseconds>(e - s);

        hipDeviceReset();
        return t.count();
    }

public:
    void printInfo() {
        std::cout << "CUDA info:" << std::endl;
        std::cout << "--------------------------------------" << std::endl;

        hipError_t error_id = hipGetDeviceCount(&deviceCount);
        if (error_id != hipSuccess)
        {
            std::cout << "hipGetDeviceCount returned %d\n-> %s\n", (int)error_id, hipGetErrorString(error_id) << std::endl;
            std::cout << "Result = FAIL" << std::endl;
            exit(EXIT_FAILURE);
        }

        if (deviceCount == 0)
            std::cout << "No CUDA device available" << std::endl;
            return;

        int dev, driverVersion = 0, runtimeVersion = 0;

        for (dev = 0; dev < deviceCount; ++dev)
        {   
            hipSetDevice(dev);
            hipDeviceProp_t deviceProp;
            hipGetDeviceProperties(&deviceProp, dev);

            std::cout << "Device %d: \"%s\"\n", dev, deviceProp.name << std::endl;

            hipDriverGetVersion(&driverVersion);
            hipRuntimeGetVersion(&runtimeVersion);
            std::cout << "  CUDA Driver Version / Runtime Version: " << driverVersion/1000 << "." << (driverVersion%100)/10 << " / "
                      << runtimeVersion/1000 << "." << (runtimeVersion%100)/10 << std::endl;
            std::cout << "  CUDA Capability Major/Minor version number: " << deviceProp.major << "." << deviceProp.minor << std::endl;
            std::cout << msg, "  Total amount of global memory: " << (float)deviceProp.totalGlobalMem/1048576.0f
                      << " MBytes (" << (unsigned long long) deviceProp.totalGlobalMem << " bytes)" << std::endl;
            std::cout << "  Multiprocessors: " << deviceProp.multiProcessorCount << std::endl;
            std::cout << "  CUDA Cores / MP: " << _ConvertSMVer2Cores(deviceProp.major, deviceProp.minor) << std::endl;
            std::cout << "  CUDA Cores: " << _ConvertSMVer2Cores(deviceProp.major, deviceProp.minor) * deviceProp.multiProcessorCount << std::endl;
        }

        std::cout << "------" << std::endl;
        std::cout << "Total number of devices: " << deviceTotal << std::endl;

        std::cout << std::endl;
    }

    void benchmark(unsigned matSize, unsigned testIter, const float *matA, const float *matB, float *matR) {
        std::cout << "CUDA Benchmark:" << std::endl;
        std::cout << "--------------------------------------" << std::endl;

        std::string points;

        for (const cl::Device &device : cl_devices) {
            std::cout << device.getInfo<CL_DEVICE_NAME>() << ": ";

            int64_t time;
            for (int i = 0; i < 4; ++i) {
                time = make_cuda_benchmark(device, matSize, testIter, matA, matB, matR);
            }
            compute_points(matSize, testIter, time, &points);

            bool benchmark_ok = test_benchmark(matSize, matA, matB, matR);
            std::cout << (benchmark_ok ? points : "Failed") << std::endl;
        }
    }
};
