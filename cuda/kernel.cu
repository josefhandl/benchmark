
#include <hip/hip_runtime.h>

__global__ void vector_add(const unsigned *testIter, const float *matA, const float *matB, float *matR) {
    int gid = blockIdx.x * blockDim.x + threadIdx.x;
    for (int i = 0; i < *testIter; ++i) {
        matR[gid] = matA[gid] + matB[gid];
    }
}
