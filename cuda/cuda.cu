#include "hip/hip_runtime.h"

#include <iostream>
#include <chrono>

#include <hip/hip_runtime.h>

#include "kernel.cu"

// https://github.com/NVIDIA/cuda-samples/blob/master/Common/hip/hip_runtime_api.h

// Beginning of GPU Architecture definitions
inline int _ConvertSMVer2Cores(int major, int minor) {
  // Defines for GPU Architecture types (using the SM version to determine
  // the # of cores per SM
  typedef struct {
    int SM;  // 0xMm (hexidecimal notation), M = SM Major version,
    // and m = SM minor version
    int Cores;
  } sSMtoCores;

  sSMtoCores nGpuArchCoresPerSM[] = {
      {0x30, 192},
      {0x32, 192},
      {0x35, 192},
      {0x37, 192},
      {0x50, 128},
      {0x52, 128},
      {0x53, 128},
      {0x60,  64},
      {0x61, 128},
      {0x62, 128},
      {0x70,  64},
      {0x72,  64},
      {0x75,  64},
      {0x80,  64},
      {0x86, 128},
      {0x87, 128},
      {0x90, 128},
      {-1, -1}};

  int index = 0;

  while (nGpuArchCoresPerSM[index].SM != -1) {
    if (nGpuArchCoresPerSM[index].SM == ((major << 4) + minor)) {
      return nGpuArchCoresPerSM[index].Cores;
    }

    index++;
  }

  // If we don't find the values, we default use the previous one
  // to run properly
  printf(
      "MapSMtoCores for SM %d.%d is undefined."
      "  Default to use %d Cores/SM\n",
      major, minor, nGpuArchCoresPerSM[index - 1].Cores);
  return nGpuArchCoresPerSM[index - 1].Cores;
}

inline const char* _ConvertSMVer2ArchName(int major, int minor) {
  // Defines for GPU Architecture types (using the SM version to determine
  // the GPU Arch name)
  typedef struct {
    int SM;  // 0xMm (hexidecimal notation), M = SM Major version,
    // and m = SM minor version
    const char* name;
  } sSMtoArchName;

  sSMtoArchName nGpuArchNameSM[] = {
      {0x30, "Kepler"},
      {0x32, "Kepler"},
      {0x35, "Kepler"},
      {0x37, "Kepler"},
      {0x50, "Maxwell"},
      {0x52, "Maxwell"},
      {0x53, "Maxwell"},
      {0x60, "Pascal"},
      {0x61, "Pascal"},
      {0x62, "Pascal"},
      {0x70, "Volta"},
      {0x72, "Xavier"},
      {0x75, "Turing"},
      {0x80, "Ampere"},
      {0x86, "Ampere"},
      {0x87, "Ampere"},
      {0x90, "Hopper"},
      {-1, "Graphics Device"}};

  int index = 0;

  while (nGpuArchNameSM[index].SM != -1) {
    if (nGpuArchNameSM[index].SM == ((major << 4) + minor)) {
      return nGpuArchNameSM[index].name;
    }

    index++;
  }

  // If we don't find the values, we default use the previous one
  // to run properly
  printf(
      "MapSMtoArchName for SM %d.%d is undefined."
      "  Default to use %s\n",
      major, minor, nGpuArchNameSM[index - 1].name);
  return nGpuArchNameSM[index - 1].name;
}

extern "C" int64_t cu_make_cuda_benchmark(const int device, unsigned matSize, unsigned testIter, const float *matA, const float *matB, float *matR) {
    hipSetDevice(device);

    auto s = std::chrono::high_resolution_clock::now();

    // allocate and set device memory
    unsigned *d_testIter;
    float *d_matA;
    float *d_matB;
    float *d_matR;
    hipMalloc((void**)&d_testIter, sizeof(unsigned));
    hipMalloc((void**)&d_matA, sizeof(float) * matSize);
    hipMalloc((void**)&d_matB, sizeof(float) * matSize);
    hipMalloc((void**)&d_matR, sizeof(float) * matSize);

    // copy data to the device
    hipMemcpy(d_testIter, &testIter, sizeof(unsigned), hipMemcpyHostToDevice);
    hipMemcpy(d_matA, matA, sizeof(float) * matSize, hipMemcpyHostToDevice);
    hipMemcpy(d_matB, matB, sizeof(float) * matSize, hipMemcpyHostToDevice);

    // run benchmark
    int blockSize = 128;
    int gridSize = matSize / blockSize;
    vector_add <<<gridSize, blockSize>>>(d_testIter, d_matA, d_matB, d_matR);

    hipMemcpy(matR, d_matR, sizeof(float) * matSize, hipMemcpyDeviceToHost);

    auto e = std::chrono::high_resolution_clock::now();
    auto t = std::chrono::duration_cast<std::chrono::microseconds>(e - s);

    hipDeviceReset();
    return t.count();
}

extern "C" void cu_printInfo(int &deviceCount) {
    hipError_t error_id = hipGetDeviceCount(&deviceCount);
    if (error_id != hipSuccess)
    {
        printf("hipGetDeviceCount returned %d\n-> %s\n", (int)error_id, hipGetErrorString(error_id));
        //std::cout << "Result = FAIL" << std::endl;
        //exit(EXIT_FAILURE);
    }

    if (deviceCount == 0)
        printf("No CUDA device available.\n");
        //return;

    int driverVersion = 0, runtimeVersion = 0;

    for (int dev = 0; dev < deviceCount; ++dev)
    {
        hipSetDevice(dev);
        hipDeviceProp_t deviceProp;
        hipGetDeviceProperties(&deviceProp, dev);

        printf("Device %d: \"%s\"\n", dev, deviceProp.name);

        // Console log
        hipDriverGetVersion(&driverVersion);
        hipRuntimeGetVersion(&runtimeVersion);
        printf("  CUDA Driver Version / Runtime Version       %d.%d / %d.%d\n", driverVersion/1000, (driverVersion%100)/10, runtimeVersion/1000, (runtimeVersion%100)/10);
        printf("  CUDA Capability Major/Minor version number: %d.%d\n", deviceProp.major, deviceProp.minor);

        //char msg[256];
        //SPRINTF(msg, "  Total amount of global memory:                 %.0f MBytes (%llu bytes)\n",
        //        (float)deviceProp.totalGlobalMem/1048576.0f, (unsigned long long) deviceProp.totalGlobalMem);
        //printf("%s", msg);
        printf("  Multiprocessors: %2d\n", deviceProp.multiProcessorCount);
        //printf("major %d, minor %d\n", deviceProp.major, deviceProp.);

        printf("  CUDA Cores/MP:   %d\n", _ConvertSMVer2Cores(deviceProp.major, deviceProp.minor));
        printf("  CUDA Cores:      %d\n", _ConvertSMVer2Cores(deviceProp.major, deviceProp.minor) * deviceProp.multiProcessorCount);

        //printf("  GPU Max Clock rate:                            %.0f MHz (%0.2f GHz)\n", deviceProp.clockRate * 1e-3f, deviceProp.clockRate * 1e-6f);
        
/*
        std::cout << "Device %d: \"%s\"\n", dev, deviceProp.name << std::endl;

        hipDriverGetVersion(&driverVersion);
        hipRuntimeGetVersion(&runtimeVersion);
        std::cout << "  CUDA Driver Version / Runtime Version: " << driverVersion/1000 << "." << (driverVersion%100)/10 << " / "
                    << runtimeVersion/1000 << "." << (runtimeVersion%100)/10 << std::endl;
        std::cout << "  CUDA Capability Major/Minor version number: " << deviceProp.major << "." << deviceProp.minor << std::endl;
        std::cout << msg, "  Total amount of global memory: " << (float)deviceProp.totalGlobalMem/1048576.0f
                    << " MBytes (" << (unsigned long long) deviceProp.totalGlobalMem << " bytes)" << std::endl;
        std::cout << "  Multiprocessors: " << deviceProp.multiProcessorCount << std::endl;
        std::cout << "  CUDA Cores / MP: " << _ConvertSMVer2Cores(deviceProp.major, deviceProp.minor) << std::endl;
        std::cout << "  CUDA Cores: " << _ConvertSMVer2Cores(deviceProp.major, deviceProp.minor) * deviceProp.multiProcessorCount << std::endl;
        */
    }

/*
    std::cout << "------" << std::endl;
    std::cout << "Total number of devices: " << deviceTotal << std::endl;

    std::cout << std::endl;
    */
}
