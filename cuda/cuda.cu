
#include <stdio.h>

#include "kernel.cu"

extern "C" int64_t cu_make_cuda_benchmark(const int device, unsigned matSize, unsigned testIter, const float *matA, const float *matB, float *matR) {
    hipSetDevice(device);

    //auto s = std::chrono::high_resolution_clock::now();

    // allocate and set device memory
    unsigned *d_testIter;
    float *d_matA;
    float *d_matB;
    float *d_matR;
    hipMalloc((void**)&d_testIter, sizeof(unsigned));
    hipMalloc((void**)&d_matA, sizeof(float) * matSize);
    hipMalloc((void**)&d_matB, sizeof(float) * matSize);
    hipMalloc((void**)&d_matR, sizeof(float) * matSize);

    // copy data to the device
    hipMemcpy(d_testIter, &testIter, sizeof(unsigned), hipMemcpyHostToDevice);
    hipMemcpy(d_matA, matA, sizeof(float) * matSize, hipMemcpyHostToDevice);
    hipMemcpy(d_matB, matB, sizeof(float) * matSize, hipMemcpyHostToDevice);
    hipMemcpy(d_matR, matR, sizeof(float) * matSize, hipMemcpyHostToDevice);

    // run benchmark
    int blockSize = 128;
    int gridSize = matSize / blockSize;
    vector_add <<<gridSize, blockSize>>>(d_testIter, d_matA, d_matB, d_matR);

    hipMemcpy(&testIter, d_testIter, sizeof(unsigned), hipMemcpyDeviceToHost);

    //auto e = std::chrono::high_resolution_clock::now();
    //auto t = std::chrono::duration_cast<std::chrono::microseconds>(e - s);

    hipDeviceReset();
    return 1; //return t.count();
}

extern "C" void cu_printInfo() {
    /*
    std::cout << "CUDA info:" << std::endl;
    std::cout << "--------------------------------------" << std::endl;
*/
    int deviceCount;
    hipError_t error_id = hipGetDeviceCount(&deviceCount);
    if (error_id != hipSuccess)
    {
        printf("hipGetDeviceCount returned %d\n-> %s\n", (int)error_id, hipGetErrorString(error_id));
        //std::cout << "Result = FAIL" << std::endl;
        exit(EXIT_FAILURE);
    }

    if (deviceCount == 0)
        printf("No CUDA device available");
        return;

    int dev, driverVersion = 0, runtimeVersion = 0;

    for (dev = 0; dev < deviceCount; ++dev)
    {
        hipSetDevice(dev);
        hipDeviceProp_t deviceProp;
        hipGetDeviceProperties(&deviceProp, dev);

        printf("Device %d: \"%s\"\n", dev, deviceProp.name);
/*
        std::cout << "Device %d: \"%s\"\n", dev, deviceProp.name << std::endl;

        hipDriverGetVersion(&driverVersion);
        hipRuntimeGetVersion(&runtimeVersion);
        std::cout << "  CUDA Driver Version / Runtime Version: " << driverVersion/1000 << "." << (driverVersion%100)/10 << " / "
                    << runtimeVersion/1000 << "." << (runtimeVersion%100)/10 << std::endl;
        std::cout << "  CUDA Capability Major/Minor version number: " << deviceProp.major << "." << deviceProp.minor << std::endl;
        std::cout << msg, "  Total amount of global memory: " << (float)deviceProp.totalGlobalMem/1048576.0f
                    << " MBytes (" << (unsigned long long) deviceProp.totalGlobalMem << " bytes)" << std::endl;
        std::cout << "  Multiprocessors: " << deviceProp.multiProcessorCount << std::endl;
        std::cout << "  CUDA Cores / MP: " << _ConvertSMVer2Cores(deviceProp.major, deviceProp.minor) << std::endl;
        std::cout << "  CUDA Cores: " << _ConvertSMVer2Cores(deviceProp.major, deviceProp.minor) * deviceProp.multiProcessorCount << std::endl;
        */
    }

/*
    std::cout << "------" << std::endl;
    std::cout << "Total number of devices: " << deviceTotal << std::endl;

    std::cout << std::endl;
    */
}
